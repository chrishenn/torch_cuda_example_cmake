#include "hip/hip_runtime.h"
/**
Authors: Christian Henn, Qianli Liao
**/

//#include <torch/types.h>
#include <torch/script.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <math.h>
#include <stdio.h>
#include <iostream>


// define this line for error checking, and put CudaCheckError() after a kernel call; I've seen little to no performance impact, though
//  the call does theoretically synchronize the device.
// #define CUDA_ERROR_CHECK

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    do{
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }

        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while(0);
#endif
    return;
}

// timing helper - works from cpu - remember to synchronize before and after kernel call
double get_nanos() {
	struct timespec ts;
	timespec_get(&ts, TIME_UTC);
	return (double)ts.tv_nsec;
}


// You can also add the __restrict__ keyword to ensure that (some types of?) memory accesses will not read off the end
// of one array into an array whose starting address is referenced by another defined variable name. Good for debugging
// but can incurr significant performance penalty.

// copies one array of ints 'input' to 'output'
__global__ void example_main(
    const int* input,
          int* output,
    const int size
){
    for (int glob_i = blockIdx.x * blockDim.x + threadIdx.x; glob_i < size; glob_i += blockDim.x * gridDim.x)
    {
        auto in_num = input[glob_i];
        output[glob_i] = in_num;
    }
}


/** 
cpu entry point for python extension.
**/
std::vector<torch::Tensor> example_call(
    torch::Tensor input
) {

    // indexing gives us advanced slicing support using the torch c++ api
    // note: .item() syntax is templated in c++ torch-api, as in:  .item<int>()
    using namespace torch::indexing;

    // set device
    auto device = input.get_device();
    hipSetDevice(device);

    // tensor allocation
    auto int_opt = torch::TensorOptions()
            .dtype(torch::kInt32)
            .layout(torch::kStrided)
            .device(torch::kCUDA, device)
            .requires_grad(false);

    auto output = torch::empty(input.size(0), int_opt);

    // calculate grid size
    // This code attempts to fit 2 cuda blocks per sm on the device, unless blocks of 256 threads each can cover the global datasize
    //  with fewer total blocks.
    // I've found this scheme to work best in limited experiments.
    // Blocks of more than 512 threads generally see performance regression on pascal and newer.
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    int n_threads = 256;
    int sms = deviceProp.multiProcessorCount;
    int full_cover = (input.size(0)-1) / n_threads + 1;
    int n_blocks = min(full_cover, 2 * sms);

    const dim3 blocks(n_blocks);
    const dim3 threads(n_threads);

    // there is a way to dispatch to a templated kernel definition, switching on a tensor dtype. Good luck finding documentation;
    //  however, frnn can dispatch to templates supporting torch.float and torch.float16 if you need that syntax.
    example_main<<<blocks, threads>>>(
        input.data_ptr<int>(),
        output.data_ptr<int>(),
        output.size(0)
    );
    CudaCheckError();

    // the std::vector of torch::Tensor 's in the c++ api will return a python-list of tensors to python
    // some c++ primitives appear to be supported.
    return {input, output};
}



